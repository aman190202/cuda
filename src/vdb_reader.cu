#include "hip/hip_runtime.h"
#include <openvdb/openvdb.h>
#include <openvdb/tools/Filter.h>
#include <iostream>
#include <vector>
#include <algorithm>
#include "light.h"




void readAndPrintVDB(const char* filename) {
    openvdb::initialize();

    openvdb::io::File file(filename);

    try {
        file.open();
        for (auto nameIter = file.beginName(); nameIter != file.endName(); ++nameIter) {
            std::cout << "Found grid: " << nameIter.gridName() << std::endl;

            openvdb::GridBase::Ptr baseGrid = file.readGrid(nameIter.gridName());
            std::cout << "Grid bbox: " << baseGrid->evalActiveVoxelBoundingBox() << std::endl;
        }
        file.close();
    } catch (const openvdb::IoError& e) {
        std::cerr << "Failed to read VDB: " << e.what() << std::endl;
    }
}



// Function to map temperature to RGB color based on Kelvin scale
__host__ __device__ vec3 temperatureToColor(float temp) {
    // Scale to Kelvin (e.g., 1000K to 7000K)
    temp = temp * 4500.0f;
    float kelvin = std::clamp(temp, 1000.0f, 7000.0f);
    
    // Normalize for interpolation (1000K to 7000K)
    float t = (kelvin - 1000.0f) / (7000.0f - 1000.0f);
    
    // Smooth interpolation for black-body color ramp
    float r = 1.0f;
    float g = std::clamp(0.0f + t * 1.2f, 0.0f, 1.0f);  // from dark red to yellow
    float b = std::clamp((t - 0.5f) * 2.0f, 0.0f, 1.0f); // blue kicks in after midpoint
    
    // Optional: Apply simple gamma correction
    float gamma = 2.2f;
    r = pow(r, 1.0f / gamma);
    g = pow(g, 1.0f / gamma);
    b = pow(b, 1.0f / gamma);
    
    return vec3(r, g, b);
}


std::vector<light> getLightsFromVDB(const char* filename, float* voxel_size, vec3* world_min, vec3* world_max) {
    std::vector<light> lights;
    openvdb::initialize();

    openvdb::io::File file(filename);
    try {
        file.open();
        
        // Find and process only the flames grid
        openvdb::GridBase::Ptr baseGrid = file.readGrid("temperature");
        openvdb::FloatGrid::Ptr floatGrid = openvdb::gridPtrCast<openvdb::FloatGrid>(baseGrid);
        
        if (floatGrid) {
            float minValue = std::numeric_limits<float>::max();
            float maxValue = std::numeric_limits<float>::lowest();
            
            // Get the bounding box in world coordinates
            openvdb::CoordBBox bbox = floatGrid->evalActiveVoxelBoundingBox();
            openvdb::Vec3d worldMin = floatGrid->indexToWorld(bbox.min());
            openvdb::Vec3d worldMax = floatGrid->indexToWorld(bbox.max());
            
            // Get and print voxel size
            openvdb::Vec3d voxelSize = floatGrid->voxelSize();
            std::cout << "Voxel size: (" << voxelSize.x() << ", " << voxelSize.y() << ", " << voxelSize.z() << ")" << std::endl;
            *voxel_size = voxelSize.x() * 2;
            *world_min = vec3(worldMin.x(), worldMin.y(), worldMin.z());
            *world_max = vec3(worldMax.x(), worldMax.y(), worldMax.z());
            
            // std::cout << "Original bounding box (world coordinates):" << std::endl;
            // std::cout << "  Min: (" << worldMin.x() << ", " << worldMin.y() << ", " << worldMin.z() << ")" << std::endl;
            // std::cout << "  Max: (" << worldMax.x() << ", " << worldMax.y() << ", " << worldMax.z() << ")" << std::endl;
            
            // Calculate the height of the bounding box
            double height = worldMax.y() - worldMin.y();
            
            // Target base position
            openvdb::Vec3d targetBase(0.0, -50.0, 0.0);
            
            // Scale factor for the bounding box
            const float scaleFactor = 2.0f;
            
            // Calculate the new height after scaling
            double scaledHeight = height * scaleFactor;
            
            // Calculate scaled min and max positions for lights
            openvdb::Vec3d scaledMin(
                targetBase.x() + (worldMin.x() - (worldMin.x() + worldMax.x()) * 0.5) * scaleFactor,
                targetBase.y(),
                targetBase.z() + (worldMin.z() - (worldMin.z() + worldMax.z()) * 0.5) * scaleFactor
            );
            
            openvdb::Vec3d scaledMax(
                targetBase.x() + (worldMax.x() - (worldMin.x() + worldMax.x()) * 0.5) * scaleFactor,
                targetBase.y() + scaledHeight,
                targetBase.z() + (worldMax.z() - (worldMin.z() + worldMax.z()) * 0.5) * scaleFactor
            );

            std::cout << "Modified bounding box for lights:" << std::endl;
            std::cout << "  Min: (" << scaledMin.x() << ", " << scaledMin.y() << ", " << scaledMin.z() << ")" << std::endl;
            std::cout << "  Max: (" << scaledMax.x() << ", " << scaledMax.y() << ", " << scaledMax.z() << ")" << std::endl;
            std::cout << "  Height: " << scaledHeight << std::endl;
            
            // Iterate through all active voxels
            for (openvdb::FloatGrid::ValueOnIter iter = floatGrid->beginValueOn(); iter; ++iter) {
                float value = iter.getValue();
                minValue = std::min(minValue, value);
                maxValue = std::max(maxValue, value);
                

                if((value * 4500) < 1000) continue;
                // Get world position of the voxel
                openvdb::Vec3d worldPos = floatGrid->indexToWorld(iter.getCoord());
                
                // Calculate relative position within the bounding box (0 to 1)
                double relY = (worldPos.y() - worldMin.y()) / height;
                
                // Calculate new position
                // X and Z are scaled and centered around targetBase
                // Y is positioned relative to the new base
                openvdb::Vec3d scaledPos(
                    targetBase.x() + (worldPos.x() - (worldMin.x() + worldMax.x()) * 0.5) * scaleFactor,
                    targetBase.y() + scaledHeight * relY,
                    targetBase.z() + (worldPos.z() - (worldMin.z() + worldMax.z()) * 0.5) * scaleFactor
                );
                
                // Create a new light
                light l;
                l.position = vec3(scaledPos.x(), scaledPos.y(), scaledPos.z());
                l.col = temperatureToColor(value);

                l.intensity = value/5; // Default intensity

                if(l.intensity < 0.05) continue;
                
                lights.push_back(l);
            }
            
            // std::cout << "Flames grid value range: [" << minValue << ", " << maxValue << "]" << std::endl;
            // std::cout << "Volume scaled by factor: " << scaleFactor << std::endl;
            // std::cout << "New base position: (" << targetBase.x() << ", " << targetBase.y() << ", " << targetBase.z() << ")" << std::endl;
        } else {
            std::cerr << "Could not find or cast flames grid" << std::endl;
        }
        
        file.close();
    } catch (const openvdb::IoError& e) {
        std::cerr << "Failed to read VDB: " << e.what() << std::endl;
    }
    
    return lights;
}



// Function to get the scaled and transformed bounding box coordinates
void getScaledBoundingBox(const char* filename, vec3& outMin, vec3& outMax) {
    openvdb::initialize();
    outMin = vec3(0.0f);
    outMax = vec3(0.0f);

    openvdb::io::File file(filename);
    try {
        file.open();
        
        // Find and process the density grid
        openvdb::GridBase::Ptr baseGrid = file.readGrid("density");
        openvdb::FloatGrid::Ptr floatGrid = openvdb::gridPtrCast<openvdb::FloatGrid>(baseGrid);
        
        if (floatGrid) {
            // Get the grid's bounding box
            openvdb::CoordBBox bbox = floatGrid->evalActiveVoxelBoundingBox();
            openvdb::Vec3d worldMin = floatGrid->indexToWorld(bbox.min());
            openvdb::Vec3d worldMax = floatGrid->indexToWorld(bbox.max());
            
            // Calculate the height of the bounding box
            double height = worldMax.y() - worldMin.y();
            
            // Target base position
            openvdb::Vec3d targetBase(0.0, -50.0, 0.0);
            
            // Scale factor for the bounding box
            const float scaleFactor = 2.0f;
            
            // Calculate the new height after scaling
            double scaledHeight = height * scaleFactor;
            
            // Calculate scaled min and max positions
            openvdb::Vec3d scaledMin(
                targetBase.x() + (worldMin.x() - (worldMin.x() + worldMax.x()) * 0.5) * scaleFactor,
                targetBase.y(),
                targetBase.z() + (worldMin.z() - (worldMin.z() + worldMax.z()) * 0.5) * scaleFactor
            );
            
            openvdb::Vec3d scaledMax(
                targetBase.x() + (worldMax.x() - (worldMin.x() + worldMax.x()) * 0.5) * scaleFactor,
                targetBase.y() + scaledHeight,
                targetBase.z() + (worldMax.z() - (worldMin.z() + worldMax.z()) * 0.5) * scaleFactor
            );
            
            // Convert to vec3
            outMin = vec3(scaledMin.x(), scaledMin.y(), scaledMin.z());
            outMax = vec3(scaledMax.x(), scaledMax.y(), scaledMax.z());
        }
        
        file.close();
    } catch (const openvdb::IoError& e) {
        std::cerr << "Failed to read VDB: " << e.what() << std::endl;
    }
}

std::vector<float> getDenseGridFromVDB(const char* vdb_file, int& nx, int& ny, int& nz) 
{
    openvdb::initialize();
    openvdb::io::File file(vdb_file);
    file.open();
    auto baseGrid = file.readGrid("density");
    auto floatGrid = openvdb::gridPtrCast<openvdb::FloatGrid>(baseGrid);

    if (!floatGrid) {
        std::cerr << "ERROR: 'density' grid not found in VDB file." << std::endl;
        exit(1);
    }

    auto bbox = floatGrid->evalActiveVoxelBoundingBox();
    if (bbox.empty()) {
        std::cerr << "ERROR: Bounding box is empty." << std::endl;
        exit(1);
    }

    auto dim = bbox.dim();
    nx = dim.x();
    ny = dim.y();
    nz = dim.z();

    std::vector<float> denseGrid(nx * ny * nz, 0.0f);

    // Iterate only active voxels
    for (auto iter = floatGrid->cbeginValueOn(); iter.test(); ++iter) {
        openvdb::Coord coord = iter.getCoord();
        openvdb::Coord local = coord - bbox.min();

        // Bounds check
        if (local.x() >= 0 && local.x() < nx &&
            local.y() >= 0 && local.y() < ny &&
            local.z() >= 0 && local.z() < nz) 
        {
            denseGrid[local.x() + local.y() * nx + local.z() * nx * ny] = iter.getValue();
        }
    }

    file.close();
    return denseGrid;
}

std::vector<float> gaussianSmooth(const std::vector<float>& grid, int nx, int ny, int nz, float sigma) {
    int kernel_radius = 1; // 3x3x3 kernel
    std::vector<float> kernel(27);
    
    // Generate Gaussian kernel
    float sum = 0.0f;
    for (int z = -kernel_radius; z <= kernel_radius; ++z) {
        for (int y = -kernel_radius; y <= kernel_radius; ++y) {
            for (int x = -kernel_radius; x <= kernel_radius; ++x) {
                float value = expf(-(x * x + y * y + z * z) / (2.0f * sigma * sigma));
                kernel[(z + kernel_radius) * 9 + (y + kernel_radius) * 3 + (x + kernel_radius)] = value;
                sum += value;
            }
        }
    }
    for (float& v : kernel) v /= sum;

    std::vector<float> smoothed(grid.size(), 0.0f);

    // Apply filter
    for (int z = 0; z < nz; ++z) {
        for (int y = 0; y < ny; ++y) {
            for (int x = 0; x < nx; ++x) {
                float accum = 0.0f;
                for (int kz = -kernel_radius; kz <= kernel_radius; ++kz) {
                    for (int ky = -kernel_radius; ky <= kernel_radius; ++ky) {
                        for (int kx = -kernel_radius; kx <= kernel_radius; ++kx) {
                            int sx = x + kx;
                            int sy = y + ky;
                            int sz = z + kz;
                            if (sx >= 0 && sx < nx && sy >= 0 && sy < ny && sz >= 0 && sz < nz) {
                                accum += grid[sx + sy * nx + sz * nx * ny] * 
                                         kernel[(kz + kernel_radius) * 9 + (ky + kernel_radius) * 3 + (kx + kernel_radius)];
                            }
                        }
                    }
                }
                smoothed[x + y * nx + z * nx * ny] = accum;
            }
        }
    }


    return smoothed;
}



std::vector<light> getTopNLightsFromVDB(
    const char* filename,
    float* voxel_size,
    vec3* world_min,
    vec3* world_max,
    size_t maxLights)
{
    std::vector<light> lights;
    openvdb::initialize();
    openvdb::io::File file(filename);
    file.open();
    auto baseGrid = file.readGrid("temperature");
    auto floatGrid = openvdb::gridPtrCast<openvdb::FloatGrid>(baseGrid);
    if (!floatGrid) { file.close(); return lights; }

    auto bbox = floatGrid->evalActiveVoxelBoundingBox();
    auto wmin = floatGrid->indexToWorld(bbox.min());
    auto wmax = floatGrid->indexToWorld(bbox.max());
    *world_min = vec3(wmin.x(), wmin.y(), wmin.z());
    *world_max = vec3(wmax.x(), wmax.y(), wmax.z());
    *voxel_size = static_cast<float>(floatGrid->voxelSize().x() * 2);

    double height = wmax.y() - wmin.y();
    const openvdb::Vec3d targetBase(0.0, -50.0, 0.0);
    const float scaleFactor = 2.0f;

    for (auto iter = floatGrid->beginValueOn(); iter; ++iter) {
        float value = iter.getValue();
        if ((value * 4500) < 1000) continue;
        auto wp = floatGrid->indexToWorld(iter.getCoord());
        double relY = (wp.y() - wmin.y()) / height;
        openvdb::Vec3d sp(
            targetBase.x() + (wp.x() - (wmin.x() + wmax.x()) * 0.5) * scaleFactor,
            targetBase.y() + height * scaleFactor * relY,
            targetBase.z() + (wp.z() - (wmin.z() + wmax.z()) * 0.5) * scaleFactor
        );
        light l;
        l.position  = vec3(sp.x(), sp.y(), sp.z());
        l.col       = temperatureToColor(value);
        l.intensity = value / 5.0f;
        if (l.intensity < 0.05f) continue;
        lights.push_back(l);
    }

    file.close();

    if (lights.size() > maxLights) {
        std::random_device rd;
        std::mt19937 gen(rd());
        std::shuffle(lights.begin(), lights.end(), gen);
        lights.resize(maxLights);
    }

    return lights;
}