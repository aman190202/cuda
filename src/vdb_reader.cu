#include "hip/hip_runtime.h"
#include <openvdb/openvdb.h>
#include <openvdb/tools/Filter.h>
#include <iostream>
#include <vector>
#include <algorithm>
#include "light.h"




void readAndPrintVDB(const char* filename) {
    openvdb::initialize();

    openvdb::io::File file(filename);

    try {
        file.open();
        for (auto nameIter = file.beginName(); nameIter != file.endName(); ++nameIter) {
            std::cout << "Found grid: " << nameIter.gridName() << std::endl;

            openvdb::GridBase::Ptr baseGrid = file.readGrid(nameIter.gridName());
            std::cout << "Grid bbox: " << baseGrid->evalActiveVoxelBoundingBox() << std::endl;
        }
        file.close();
    } catch (const openvdb::IoError& e) {
        std::cerr << "Failed to read VDB: " << e.what() << std::endl;
    }
}



// Function to map temperature to RGB color based on Kelvin scale
__host__ __device__ vec3 temperatureToColor(float temp) {
    // Scale to Kelvin (e.g., 1000K to 7000K)
    temp = temp * 4500.0f;
    float kelvin = std::clamp(temp, 1000.0f, 7000.0f);
    
    // Normalize for interpolation (1000K to 7000K)
    float t = (kelvin - 1000.0f) / (7000.0f - 1000.0f);
    
    // Smooth interpolation for black-body color ramp
    float r = 1.0f;
    float g = std::clamp(0.0f + t * 1.2f, 0.0f, 1.0f);  // from dark red to yellow
    float b = std::clamp((t - 0.5f) * 2.0f, 0.0f, 1.0f); // blue kicks in after midpoint
    
    // Optional: Apply simple gamma correction
    float gamma = 2.2f;
    r = pow(r, 1.0f / gamma);
    g = pow(g, 1.0f / gamma);
    b = pow(b, 1.0f / gamma);
    
    return vec3(r, g, b);
}


std::vector<light> getLightsFromVDB(const char* filename) {
    std::vector<light> lights;
    openvdb::initialize();

    openvdb::io::File file(filename);
    try {
        file.open();
        
        // Find and process only the flames grid
        openvdb::GridBase::Ptr baseGrid = file.readGrid("temperature");
        openvdb::FloatGrid::Ptr floatGrid = openvdb::gridPtrCast<openvdb::FloatGrid>(baseGrid);
        
        if (floatGrid) {
            float minValue = std::numeric_limits<float>::max();
            float maxValue = std::numeric_limits<float>::lowest();
            
            // Get the bounding box in world coordinates
            openvdb::CoordBBox bbox = floatGrid->evalActiveVoxelBoundingBox();
            openvdb::Vec3d worldMin = floatGrid->indexToWorld(bbox.min());
            openvdb::Vec3d worldMax = floatGrid->indexToWorld(bbox.max());
            
            std::cout << "Original bounding box (world coordinates):" << std::endl;
            std::cout << "  Min: (" << worldMin.x() << ", " << worldMin.y() << ", " << worldMin.z() << ")" << std::endl;
            std::cout << "  Max: (" << worldMax.x() << ", " << worldMax.y() << ", " << worldMax.z() << ")" << std::endl;
            
            // Calculate the height of the bounding box
            double height = worldMax.y() - worldMin.y();
            
            // Target base position
            openvdb::Vec3d targetBase(0.0, -50.0, 0.0);
            
            // Scale factor for the bounding box
            const float scaleFactor = 2.0f;
            
            // Calculate the new height after scaling
            double scaledHeight = height * scaleFactor;
            
            // Iterate through all active voxels
            for (openvdb::FloatGrid::ValueOnIter iter = floatGrid->beginValueOn(); iter; ++iter) {
                float value = iter.getValue();
                minValue = std::min(minValue, value);
                maxValue = std::max(maxValue, value);
                

                if((value * 4500) < 1000) continue;
                // Get world position of the voxel
                openvdb::Vec3d worldPos = floatGrid->indexToWorld(iter.getCoord());
                
                // Calculate relative position within the bounding box (0 to 1)
                double relY = (worldPos.y() - worldMin.y()) / height;
                
                // Calculate new position
                // X and Z are scaled and centered around targetBase
                // Y is positioned relative to the new base
                openvdb::Vec3d scaledPos(
                    targetBase.x() + (worldPos.x() - (worldMin.x() + worldMax.x()) * 0.5) * scaleFactor,
                    targetBase.y() + scaledHeight * relY,
                    targetBase.z() + (worldPos.z() - (worldMin.z() + worldMax.z()) * 0.5) * scaleFactor
                );
                
                // Create a new light
                light l;
                l.position = vec3(scaledPos.x(), scaledPos.y(), scaledPos.z());
                l.col = temperatureToColor(value);

                l.intensity = value/5; // Default intensity

                if(l.intensity < 0.05) continue;
                
                lights.push_back(l);
            }
            
            // std::cout << "Flames grid value range: [" << minValue << ", " << maxValue << "]" << std::endl;
            // std::cout << "Volume scaled by factor: " << scaleFactor << std::endl;
            // std::cout << "New base position: (" << targetBase.x() << ", " << targetBase.y() << ", " << targetBase.z() << ")" << std::endl;
        } else {
            std::cerr << "Could not find or cast flames grid" << std::endl;
        }
        
        file.close();
    } catch (const openvdb::IoError& e) {
        std::cerr << "Failed to read VDB: " << e.what() << std::endl;
    }
    
    return lights;
}



// Function to get the scaled and transformed bounding box coordinates
void getScaledBoundingBox(const char* filename, vec3& outMin, vec3& outMax) {
    openvdb::initialize();
    outMin = vec3(0.0f);
    outMax = vec3(0.0f);

    openvdb::io::File file(filename);
    try {
        file.open();
        
        // Find and process the density grid
        openvdb::GridBase::Ptr baseGrid = file.readGrid("density");
        openvdb::FloatGrid::Ptr floatGrid = openvdb::gridPtrCast<openvdb::FloatGrid>(baseGrid);
        
        if (floatGrid) {
            // Get the grid's bounding box
            openvdb::CoordBBox bbox = floatGrid->evalActiveVoxelBoundingBox();
            openvdb::Vec3d worldMin = floatGrid->indexToWorld(bbox.min());
            openvdb::Vec3d worldMax = floatGrid->indexToWorld(bbox.max());
            
            // Calculate the height of the bounding box
            double height = worldMax.y() - worldMin.y();
            
            // Target base position
            openvdb::Vec3d targetBase(0.0, -50.0, 0.0);
            
            // Scale factor for the bounding box
            const float scaleFactor = 2.0f;
            
            // Calculate the new height after scaling
            double scaledHeight = height * scaleFactor;
            
            // Calculate scaled min and max positions
            openvdb::Vec3d scaledMin(
                targetBase.x() + (worldMin.x() - (worldMin.x() + worldMax.x()) * 0.5) * scaleFactor,
                targetBase.y(),
                targetBase.z() + (worldMin.z() - (worldMin.z() + worldMax.z()) * 0.5) * scaleFactor
            );
            
            openvdb::Vec3d scaledMax(
                targetBase.x() + (worldMax.x() - (worldMin.x() + worldMax.x()) * 0.5) * scaleFactor,
                targetBase.y() + scaledHeight,
                targetBase.z() + (worldMax.z() - (worldMin.z() + worldMax.z()) * 0.5) * scaleFactor
            );
            
            // Convert to vec3
            outMin = vec3(scaledMin.x(), scaledMin.y(), scaledMin.z());
            outMax = vec3(scaledMax.x(), scaledMax.y(), scaledMax.z());
        }
        
        file.close();
    } catch (const openvdb::IoError& e) {
        std::cerr << "Failed to read VDB: " << e.what() << std::endl;
    }
}
/*
std::vector<float> getDenseGridFromVDB(const char* vdb_file, int& nx, int& ny, int& nz) 
{
    openvdb::initialize();
    openvdb::io::File file(vdb_file);
    file.open();
    auto baseGrid = file.readGrid("density");
    auto floatGrid = openvdb::gridPtrCast<openvdb::FloatGrid>(baseGrid);

    // Null grid check
    if (!floatGrid) {
        std::cerr << "ERROR: 'density' grid not found in VDB file. Aborting." << std::endl;
        exit(1);
    }

    auto bbox = floatGrid->evalActiveVoxelBoundingBox();

    // Empty bbox check
    if (bbox.empty()) {
        std::cerr << "ERROR: 'density' grid has empty bounding box (no active voxels). Aborting." << std::endl;
        exit(1);
    }

    auto dim = bbox.dim();
    nx = dim.x();
    ny = dim.y();
    nz = dim.z();

    // Sanity check
    if (nx <= 0 || ny <= 0 || nz <= 0) {
        std::cerr << "ERROR: Invalid grid dimensions: " << nx << " " << ny << " " << nz << std::endl;
        exit(1);
    }

    std::vector<float> denseGrid(nx * ny * nz, 0.0f);
    auto accessor = floatGrid->getAccessor();
    for (int z = 0; z < nz; ++z) {
        for (int y = 0; y < ny; ++y) {
            for (int x = 0; x < nx; ++x) {
                auto coord = bbox.min().offsetBy(x, y, z);
                denseGrid[x + y * nx + z * nx * ny] = accessor.getValue(coord);
            }
        }
    }

    file.close();
    return denseGrid;
}
*/

std::vector<float> getDenseGridFromVDB(const char* vdb_file, int& nx, int& ny, int& nz) 
{
    openvdb::initialize();
    openvdb::io::File file(vdb_file);
    file.open();
    auto baseGrid = file.readGrid("density");
    auto floatGrid = openvdb::gridPtrCast<openvdb::FloatGrid>(baseGrid);

    if (!floatGrid) {
        std::cerr << "ERROR: 'density' grid not found in VDB file." << std::endl;
        exit(1);
    }

    auto bbox = floatGrid->evalActiveVoxelBoundingBox();
    if (bbox.empty()) {
        std::cerr << "ERROR: Bounding box is empty." << std::endl;
        exit(1);
    }

    auto dim = bbox.dim();
    nx = dim.x();
    ny = dim.y();
    nz = dim.z();

    std::vector<float> denseGrid(nx * ny * nz, 0.0f);

    // Iterate only active voxels
    for (auto iter = floatGrid->cbeginValueOn(); iter.test(); ++iter) {
        openvdb::Coord coord = iter.getCoord();
        openvdb::Coord local = coord - bbox.min();

        // Bounds check
        if (local.x() >= 0 && local.x() < nx &&
            local.y() >= 0 && local.y() < ny &&
            local.z() >= 0 && local.z() < nz) 
        {
            denseGrid[local.x() + local.y() * nx + local.z() * nx * ny] = iter.getValue();
        }
    }

    file.close();
    return denseGrid;
}


// std::vector<float> getDenseGridFromVDB(const char* vdb_file, int& nx, int& ny, int& nz) 
// {
//     openvdb::initialize();
//     openvdb::io::File file(vdb_file);
//     file.open();

//     auto baseGrid = file.readGrid("density");
//     auto floatGrid = openvdb::gridPtrCast<openvdb::FloatGrid>(baseGrid);

//     // Null grid check
//     if (!floatGrid) {
//         std::cerr << "ERROR: 'density' grid not found in VDB file. Aborting." << std::endl;
//         exit(1);
//     }

//     // --------->>> ADD HERE <<<<<--------
//     // Apply Gaussian blur to smooth density grid (important for mushroom shape)
//     openvdb::tools::GaussianFilter<openvdb::FloatGrid> filter(*floatGrid);
//     filter.setSigma(1.0);  // Sigma controls smoothness
//     filter.apply();
//     // -----------------------------------

//     auto bbox = floatGrid->evalActiveVoxelBoundingBox();
//     auto dim = bbox.dim();
//     nx = dim.x();
//     ny = dim.y();
//     nz = dim.z();

//     std::vector<float> denseGrid(nx * ny * nz, 0.0f);
//     auto accessor = floatGrid->getAccessor();
//     for (int z = 0; z < nz; ++z) {
//         for (int y = 0; y < ny; ++y) {
//             for (int x = 0; x < nx; ++x) {
//                 auto coord = bbox.min().offsetBy(x, y, z);
//                 denseGrid[x + y * nx + z * nx * ny] = accessor.getValue(coord);
//             }
//         }
//     }

//     file.close();
//     return denseGrid;
// }